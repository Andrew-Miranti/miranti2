#include "hip/hip_runtime.h"
#include <string.h>	

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <omp.h>

#include "tbb/tick_count.h" 
using tbb::tick_count;

char* map_file(char *filename, int *length_out) 
{
	struct stat file_stat;
	int fd = open(filename, O_RDONLY);
	if (fd == -1) 
	{
		printf("failed to open file: %s\n", filename); 
		exit(1);
	}
	if (fstat(fd, &file_stat) != 0) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}
	off_t length = file_stat.st_size;
	void *file = mmap(0, length, PROT_WRITE, MAP_PRIVATE, fd, 0);
	if (file == (void *)-1) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}

	*length_out = length;
	return (char *)file;
}

#define HISTOGRAM_SIZE 256

__global__ void countLetters(char * file, unsigned * allHistogram, int length, int total) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned * histogram = allHistogram + index * HISTOGRAM_SIZE;
    int startIndex = index * length / total;
    int endIndex = (index+1) * length / total;
    for (int i = startIndex; i < endIndex; ++i) {
        histogram[file[i]]++;
    }
}


int main(int argc, char *argv[]) 
{
	int length = 0;
	bool print = false;
	if (strncmp(argv[1], "-print", 7) == 0) 
	{
		print = true;
		argv = &argv[1];
	}
	char *file = map_file(argv[1], &length);
	unsigned histogram[HISTOGRAM_SIZE] = {0};

	tick_count start = tick_count::now();

    char * deviceFile = NULL;
    hipMalloc((void**)&deviceFile, length);
    hipMemcpy(deviceFile, file, length, hipMemcpyHostToDevice);
    unsigned * histograms = NULL;
    int numBlocks = 4;
    int numThreads = 4;
    int totalThreads = numBlocks * numThreads;
    size_t allHistogramSize = sizeof(unsigned) * totalThreads * HISTOGRAM_SIZE;
    hipMalloc((void**)&histograms, allHistogramSize);
    hipMemset(histograms, 0, allHistogramSize);
    countLetters<<<numBlocks, numThreads>>>(deviceFile, histograms, length, totalThreads);
    unsigned * hostHistograms = (unsigned*)malloc(allHistogramSize);
    memset(hostHistograms, 0, allHistogramSize);

    hipMemcpy(hostHistograms, histograms, allHistogramSize, hipMemcpyDeviceToHost);
    hipFree(deviceFile);
    hipFree(histograms);

    for (int j = 0; j < totalThreads; ++j) {
        for (int i = 0; i < HISTOGRAM_SIZE; ++i) {
            histogram[i] += hostHistograms[HISTOGRAM_SIZE * j + i];
        }  
    }
    free(hostHistograms);

	tick_count end = tick_count::now();

	printf("time = %f seconds\n", (end - start).seconds());  

	if (print) 
	{
		for (int i = 0 ; i < 128 ; i ++) 
		{
			if (histogram[i] != 0) 
			{
				printf("%c (%d): %d\n", i, i, histogram[i]);
			}
		}
	}
}
